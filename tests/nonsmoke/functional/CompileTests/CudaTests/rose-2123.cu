
#include <hip/hip_runtime.h>

template <class CudaFn>
__global__
void launcher(CudaFn fn) {
  fn();
}

int main() {
//  launcher<<< 1, 1>>>( [] () -> void {} );
  launcher<<< 1, 1>>>( [] __device__ () -> void {} );
  return 0;
}
