// compile with: nvcc --expt-extended-lambda -std c++11 -c deviceLambda.cu

#include <hip/hip_runtime.h>


template <class CudaFn>
__global__
void launcher(CudaFn fn)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  fn(idx);
}

int main()
{
  static const int sz = 256;
  static const int bl = 32;

  double* matrix = nullptr;

  hipMalloc(&matrix, sz*sz*sizeof(double));

  dim3   dimBlock(bl, bl);
  dim3   dimGrid((sz+bl-1)/bl, (sz+bl-1)/bl);

  // device lambda
  launcher<<<dimGrid, dimBlock>>>( [matrix] __device__ (int idx) -> void
                                   {
                                     matrix[idx] = 0;
                                   }
                                 );
  return 0;
}
