#include <hip/hip_runtime.h>


__global__
void
__launch_bounds__(128, 2)
init(double *x)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  x[idx] = 0;
}

int main()
{
  static const int sz = 256;
  static const int bl = 32;

  double* matrix = nullptr;

  hipMalloc(&matrix, sz*sz*sizeof(double));

  dim3   dimBlock(bl, bl);
  dim3   dimGrid((sz+bl-1)/bl, (sz+bl-1)/bl);
  init<<<dimGrid, dimBlock>>>(matrix);

  return 0;
}
