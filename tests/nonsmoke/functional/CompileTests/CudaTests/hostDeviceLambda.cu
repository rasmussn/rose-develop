// compile with: nvcc --expt-extended-lambda -std c++11 -c hostDdeviceLambda.cu

#include <hip/hip_runtime.h>


template <class CudaFn>
__global__
void launcher(CudaFn fn)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  fn(idx);
}

template <class Fn>
void run(int sz, Fn fn)
{
  // run large matrices on GPU
  if (sz*sz >= 256)
  {
    static const int bl = 32;

    dim3   dimBlock(bl, bl);
    dim3   dimGrid((sz+bl-1)/bl, (sz+bl-1)/bl);

    launcher<<<dimGrid, dimBlock>>>(fn);
    return;
  }

  // and small ones sequentially
  for (int i = 0; i < sz*sz; ++i)
    fn(i);
}

int main()
{
  static const int sz = 256;

  double* matrix = nullptr;

  hipMallocManaged(&matrix, sz*sz*sizeof(double));

  // host device lambda
  run( sz, [matrix] __host__ __device__ (int idx) -> void
       {
         matrix[idx] = 0;
       }
     );
  return 0;
}
